#include "simple.cu"
#include <iostream>
#include <hip/hip_runtime.h>


int main() {
    const int rows = 4;
    const int cols = 4;
    const int size = rows * cols * sizeof(float);

    float h_A[rows * cols], h_B[rows * cols], h_C[rows * cols];
    for (int i = 0; i < rows * cols; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Result matrix C:\n";
    for (int i = 0; i < rows * cols; ++i) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % cols == 0) std::cout << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
